#include <iostream>
#include "kernels.cuh"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#define CUDA_CHECK {                                                                       \
 hipError_t e=hipGetLastError();                                                         \
 if(e!=hipSuccess) {                                                                      \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
   exit(0);                                                                                \
 }                                                                                         \
}
/* important init_mat 100
int activator_radii[SCALE_COUNT]    = {4,8,16,32,64,128,256,384,512,768};
int inhibitor_radii[SCALE_COUNT]    = {8,16,32,64,128,256,512,768,1024,1536};
int variation_radii[SCALE_COUNT]    = {1,1,1,1,1,1,1,1,1,1};
float modifiers_values[SCALE_COUNT] = {0.002,0.002,0.002,0.002,0.002,0.002,0.002,0.002,0.002,0.002};
int calculation_vector[SCALE_COUNT] = {1,1,1,1,1,1,1,1,1};
*/
int activator_radii[SCALE_COUNT]    = {4,16,64,256,1024};
int inhibitor_radii[SCALE_COUNT]    = {8,32,128,512,2048};
//int activator_radii[SCALE_COUNT]    = {64,128,256,512,1024};
//int inhibitor_radii[SCALE_COUNT]    = {128,256,512,1024,2048};
int variation_radii[SCALE_COUNT]    = {1,1,1,1,1};
float modifiers_values[SCALE_COUNT] = {0.002,0.002,0.002,0.002,0.002};
int calculation_vector[SCALE_COUNT] = {1,1,1,1,1};
/*
int activator_radii[SCALE_COUNT]    = {4,8,16,32,64,128,256};
int inhibitor_radii[SCALE_COUNT]    = {8,16,32,64,128,256,512};
int variation_radii[SCALE_COUNT]    = {1,1,1,1,1,1,1};
float modifiers_values[SCALE_COUNT] = {0.02,0.02,0.02,0.02,0.02,0.02,0.02};
int calculation_vector[SCALE_COUNT] = {0,0,0,0,0,0};
*/
/*
unsigned char color_map[SCALE_COUNT * 4] = {
    255,0,  0,  255,
    0,  255,0,  255,
    0,  0,  255,255,
    255,0,  0,  255,
    0,  255,0,  255,
    0,  0,  255,255,
    255,0,  0,  255,
    0,  255,0,  255,
    0,  0,  255,255,
    255,0,  0,  255,
    0,  255,0,  255,
    0,  0,  255,255,
    255,0,  0,  255,
    0,  255,0,  255,
    0,  0,  255,255,
    255,0,  0,  255,
};*/
/*
unsigned char color_map[SCALE_COUNT * 4] = {
    232,  215,  241,  255,  
    255,0,  0,  255,  
    48, 31,71,  255,  
    188,96,255,255,  
    255,255,0,  255,  
    255,0,  255,255,  
    0,  255,255,255,  
    13,19,33,255,  
    0,  0,  0,  255,
    161,103,165, 255,
    0,  255,0,  255,
    255,255,0,  255,
    74,48,108,255,
    255,0,  255,255,
    211,188,204,255,
    232,  215,  241,255,
};
*/
/*
unsigned char color_map[SCALE_COUNT * 4] = {
    255,77,77,0,
    255,165,0,0,
    255,255,0,0,
    127,255,0,0,
    0,255,0,0,
    0,255,127,0,
    0,255,255,0,
    0,127,255,0,
    0,0,255,0,
    127,0,255,0,
    255,0,255,0,
    255,0,127,0,
    139,69,19,0,
    128,128,128,0,
    0,191,255,0,
    255,20,147,0,
};
*/

/* //those look nice
unsigned char color_map[SCALE_COUNT * 4] = {
    248, 201, 119, 0,
    63, 136, 143, 0,
    198, 82, 69, 0,
    117, 183, 138, 0,
    235, 151, 78, 0,
    147, 118, 177, 0,
    87, 171, 191, 0,
    217, 127, 132, 0,
    168, 205, 136, 0,
};*/
//emma's

/*
unsigned char color_map[SCALE_COUNT * 4] = {
    227, 215, 255, 0,
    175, 162, 255, 0,
    236, 220, 11, 0,
    244, 127, 59, 0,
    209, 48, 201, 0,
    227, 215, 255, 0,
    175, 162, 255, 0,
    236, 220, 11, 0,
    244, 127, 59, 0,
    209, 48, 201, 0,
};
*/
unsigned char color_map[SCALE_COUNT * 4] = {
    //idk 0, 128, 128, 0, 255, 107, 107, 0, 143, 182, 140, 0, 107, 62, 131, 0, 255, 209, 102, 0
    0, 102, 204, 0, 0, 128, 0, 0, 153, 102, 204, 0, 64, 64, 64, 0, 150, 113, 23, 0
    //this 227, 215, 255, 0, 175, 162, 255, 0, 236, 220, 11, 0, 244, 127, 59, 0, 209, 48, 201, 0,
};
/*
unsigned char color_map[SCALE_COUNT * 4] = {
    220, 70, 89, 0,
    123, 123, 234, 0,
    255, 105, 180, 0,
    79, 129, 189, 0,
    0, 128, 128, 0,
    255, 128, 0, 0,
    //147, 112, 219, 0,
    //40, 160, 120, 0,
    156, 175, 7, 0,
};
*/
#define SIZE_REAL (DIM * DIM)
#define SIZE_COMP (DIM * (DIM / 2 + 1))
int activator_counts[SCALE_COUNT] = {0};
int inhibitor_counts[SCALE_COUNT] = {0};
int variation_counts[SCALE_COUNT] = {0};

float* d_modifiers_values;
int* d_calculation_vector;
unsigned char *d_color_map;
unsigned char *d_tex_data;

float* d_activators;
float* d_inhibitors;
float* d_variations;
int* d_MSTPcolors;
float* d_MSTPvalues;

hipfftHandle planR2C;
hipfftHandle planC2R;

hipfftComplex* d_activators_kernels;
hipfftComplex* d_inhibitors_kernels;
hipfftComplex* d_variations_kernels;
hipfftComplex* d_memory_complex;
hipfftComplex* d_MSTPvalues_complex;



void mem_init(){
    hipMalloc(&d_tex_data, SIZE_REAL * sizeof(unsigned char) * 4);

    hipMalloc(&d_calculation_vector, SCALE_COUNT * sizeof(int));
    hipMalloc(&d_modifiers_values, SCALE_COUNT * sizeof(float));
    hipMalloc(&d_color_map, SCALE_COUNT * sizeof(unsigned char) * 4);
    hipMalloc(&d_memory_complex, SIZE_COMP * sizeof(hipfftComplex));
    hipMalloc(&d_activators_kernels, SIZE_COMP * SCALE_COUNT * sizeof(hipfftComplex));
    hipMalloc(&d_inhibitors_kernels, SIZE_COMP * SCALE_COUNT * sizeof(hipfftComplex));
    hipMalloc(&d_variations_kernels, SIZE_COMP * SCALE_COUNT * sizeof(hipfftComplex));

    hipMalloc(&d_activators, SIZE_REAL * SCALE_COUNT * sizeof(float));
    hipMalloc(&d_inhibitors, SIZE_REAL * SCALE_COUNT * sizeof(float));
    hipMalloc(&d_variations, SIZE_REAL * SCALE_COUNT * sizeof(float));
    hipMalloc(&d_MSTPvalues_complex, SIZE_COMP * sizeof(hipfftComplex));
    hipMalloc(&d_MSTPcolors, SIZE_REAL * sizeof(unsigned char) * 4);
    hipMalloc(&d_MSTPvalues, SIZE_REAL * sizeof(float));

    hipfftPlan2d(&planC2R, DIM, DIM, HIPFFT_C2R);
    hipfftPlan2d(&planR2C, DIM, DIM, HIPFFT_R2C);

}
void mem_free(){

    hipFree(d_tex_data);
    hipFree(d_calculation_vector);
    hipFree(d_modifiers_values);
    hipFree(d_color_map);
    hipFree(d_memory_complex);
    hipFree(d_activators_kernels);
    hipFree(d_inhibitors_kernels);
    hipFree(d_variations_kernels);
    hipFree(d_activators);
    hipFree(d_inhibitors);
    hipFree(d_variations);
    hipFree(d_MSTPvalues_complex);
    hipFree(d_MSTPvalues);
    hipFree(d_MSTPcolors);
    hipfftDestroy(planC2R);
    hipfftDestroy(planR2C);

}
void val_init(){
    hipMemcpy(d_calculation_vector, calculation_vector, SCALE_COUNT * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_modifiers_values, modifiers_values, SCALE_COUNT * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_color_map, color_map, SCALE_COUNT * sizeof(unsigned char) * 4, hipMemcpyHostToDevice);
    int block_size = 256;
    dim3 grid(block_size, block_size);
    dim3 block(DIM / block_size, DIM / block_size);
    for(int i = 0; i < SCALE_COUNT; i++){
        init_mat<<<grid, block>>>(d_activators + i * SIZE_REAL, activator_radii[i], DIM, DIM);
        hipfftExecR2C(planR2C, d_activators + i * SIZE_REAL, d_activators_kernels + i * SIZE_COMP);
        init_mat<<<grid, block>>>(d_inhibitors + i * SIZE_REAL, inhibitor_radii[i], DIM, DIM);
        hipfftExecR2C(planR2C, d_inhibitors + i * SIZE_REAL, d_inhibitors_kernels + i * SIZE_COMP);
        init_mat<<<grid, block>>>(d_variations + i * SIZE_REAL, variation_radii[i], DIM, DIM);
        hipfftExecR2C(planR2C, d_variations + i * SIZE_REAL, d_variations_kernels + i * SIZE_COMP);
    }
    init_mat<<<grid, block>>>(d_MSTPvalues, 100, DIM, DIM);
}
void val_compute(){
    int block_size = 256;
    dim3 grid(block_size, block_size);
    dim3 block(DIM / block_size, DIM / block_size);
    hipfftExecR2C(planR2C, d_MSTPvalues, d_MSTPvalues_complex);

    for(int s = 0; s < SCALE_COUNT; s++){
        if(calculation_vector[s]){

            multiply_mat<<<grid, block>>>(d_memory_complex, d_MSTPvalues_complex, d_activators_kernels + s * SIZE_COMP, DIM, DIM / 2 + 1);
            hipfftExecC2R(planC2R, d_memory_complex, d_activators + s * SIZE_REAL);
            divide_val<<<grid, block>>>(d_activators + s * SIZE_REAL, activator_radii[s] * activator_radii[s], DIM, DIM);

            multiply_mat<<<grid, block>>>(d_memory_complex, d_MSTPvalues_complex, d_inhibitors_kernels + s * SIZE_COMP, DIM, DIM / 2 + 1);
            hipfftExecC2R(planC2R, d_memory_complex, d_inhibitors + s * SIZE_REAL);
            divide_val<<<grid, block>>>(d_inhibitors + s * SIZE_REAL, inhibitor_radii[s] * inhibitor_radii[s], DIM, DIM);

            subtract_abs_mat<<<grid, block>>>(d_variations + s * SIZE_REAL, d_inhibitors + s * SIZE_REAL, d_activators + s * SIZE_REAL, DIM, DIM);
            hipfftExecR2C(planR2C, d_variations + s * SIZE_REAL, d_memory_complex);
            multiply_mat<<<grid, block>>>(d_memory_complex, d_variations_kernels + s * SIZE_COMP, DIM, DIM / 2 + 1);
            hipfftExecC2R(planC2R, d_memory_complex, d_variations + s * SIZE_REAL);
            divide_val<<<grid, block>>>(d_variations + s * SIZE_REAL, variation_radii[s] * variation_radii[s], DIM, DIM);
        }
    }

    calculate_smallest_scale<<<grid, block>>>(d_MSTPcolors, d_variations, d_calculation_vector, DIM, DIM);
    calculate_MSTP<<<grid, block>>>(d_MSTPvalues, d_MSTPcolors, d_activators, d_inhibitors, d_modifiers_values, d_calculation_vector, DIM, DIM);
    float max_val, min_val;
    thrust::pair<float *, float *> result = thrust::minmax_element(thrust::device, d_MSTPvalues, d_MSTPvalues + SIZE_REAL);
    hipMemcpy(&min_val, result.first, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&max_val, result.second, sizeof(float), hipMemcpyDeviceToHost);
    normalize_mat<<<grid, block>>>(d_MSTPvalues, min_val, max_val, DIM, DIM);

    calculate_texture<<<grid, block>>>(d_tex_data, d_color_map, d_MSTPcolors, d_MSTPvalues, DIM, DIM);
}

int main()
{

    mem_init();
    val_init();


    for(int i = 0; i < 1280; i++){
        printf("%d\n", i);
        CUDA_CHECK;
        val_compute();  
    }

    float* h_tex_data = (float*)malloc(sizeof(unsigned char) * DIM * DIM * 4);
    hipMemcpy(h_tex_data, d_tex_data, sizeof(unsigned char) * DIM * DIM * 4, hipMemcpyDeviceToHost);
    stbi_write_png("image.png", DIM, DIM, 4, h_tex_data, DIM * 4);

    free(h_tex_data);
    mem_free();
    CUDA_CHECK;
    return 0;
}